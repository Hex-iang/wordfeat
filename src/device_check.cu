#include <utils.h>

void Initialization(int &argc, char * argv[])
{
  // Google logging
  ::google::InitGoogleLogging(argv[0]);
  // Provide a backtrace on failure
  ::google::InstallFailureSignalHandler();

#ifndef GFLAGS_GFLAGS_H_
  namespace gflags = google;
#endif 

  gflags::ParseCommandLineFlags(&argc, &argv, true);

}

int main(int argc, char* argv[])
{
  Initialization(argc, argv);

  int deviceCount = 0;
  hipGetDeviceCount(& deviceCount);
  LOG(INFO) << "CUDA Device count: " << deviceCount << ".";

  wfeatTime_start(GPU, "Getting GPU properties");

  for( int devNum = 0; devNum < deviceCount; devNum++ ) 
  {
    hipDeviceProp_t deviceProp; 

    hipGetDeviceProperties(&deviceProp, devNum);
       
    if (devNum == 0) {
      if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
        LOG(INFO) << "No CUDA GPU has been detected.";
      } else if (deviceCount == 1) {
        LOG(INFO) << "There is 1 device supporting CUDA";
      } else {
        LOG(INFO) << "There are " << deviceCount << " devices supporting CUDA";
      }   
    }
    
    // output corresponding information about devise
    LOG(INFO) << "Device " <<  devNum <<  " name: " <<  deviceProp.name;
    LOG(INFO) << " Computational Capabilities: " <<  deviceProp.major << "." << deviceProp.minor;
    LOG(INFO) << " Maximum global memory size: " <<  deviceProp.totalGlobalMem;
    LOG(INFO) << " Maximum constant memory size: ", deviceProp.totalConstMem;
    LOG(INFO) << " Maximum shared memory size per block: " << deviceProp.sharedMemPerBlock;
    LOG(INFO) << " Maximum threads per block: " << deviceProp.maxThreadsPerBlock;
    LOG(INFO) << " Maximum block dimensions: " << deviceProp.maxThreadsDim[0] << " x " 
                                               << deviceProp.maxThreadsDim[1] << " x " 
                                               << deviceProp.maxThreadsDim[2]; 
    LOG(INFO) << " Maximum grid dimensions: "  << deviceProp.maxGridSize[0] << " x "
                                               << deviceProp.maxGridSize[1] << " x "
                                               << deviceProp.maxGridSize[2];
    LOG(INFO) << " Warp size: " << deviceProp.warpSize;
  }
  
  wfeatTime_stop(GPU, "Getting GPU properties");
}
